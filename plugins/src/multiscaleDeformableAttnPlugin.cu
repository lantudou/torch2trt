#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <numeric>
#include <stdexcept>

#include "multiscaleDeformableAttnPlugin.h"
#include "multiscaleDeformableAttn.h"

#include "NvInfer.h"
#include <cassert>
#include <iostream>

using namespace nvinfer1;
namespace torch2trt_plugins {
    
MultiscaleDeformableAttnPlugin::MultiscaleDeformableAttnPlugin()
{
}

MultiscaleDeformableAttnPlugin::MultiscaleDeformableAttnPlugin(void const* data, size_t length)
{
}

nvinfer1::IPluginV2DynamicExt* MultiscaleDeformableAttnPlugin::clone() const PLUGIN_NOEXCEPT
{
    try
    {
        MultiscaleDeformableAttnPlugin* plugin = new MultiscaleDeformableAttnPlugin();
        plugin->setPluginNamespace(getPluginNamespace());
        return plugin;
    }
    catch (const std::exception& e)
    {
        std::cerr << e.what() << std::endl;
//        return nullptr;
    }
    return nullptr;
}

nvinfer1::DimsExprs MultiscaleDeformableAttnPlugin::getOutputDimensions(int32_t outputIndex,
    nvinfer1::DimsExprs const* inputs, int32_t nbInputs, nvinfer1::IExprBuilder& exprBuilder) PLUGIN_NOEXCEPT
{
    nvinfer1::DimsExprs ret;
    ret.nbDims = 4;
    ret.d[0] = inputs[0].d[0];
    ret.d[1] = inputs[3].d[1];
    ret.d[2] = inputs[0].d[2];
    ret.d[3] = inputs[0].d[3];

    return ret;
}

bool MultiscaleDeformableAttnPlugin::supportsFormatCombination(
    int32_t pos, nvinfer1::PluginTensorDesc const* inOut, int32_t nbInputs, int32_t nbOutputs) PLUGIN_NOEXCEPT
{

    assert(nbInputs == 5 && "nbInputs not equal 5");
    assert(nbOutputs == 1 && "nbOutputs not equal 1");


    if(nbInputs != 5)return false;

    if(nbOutputs != 1)return false;

    if (inOut[pos].format == nvinfer1::TensorFormat::kLINEAR)
    {
        if ((pos == 1) || (pos == 2))
        {
            return (inOut[pos].type == nvinfer1::DataType::kINT32);
        }
        else
        {
            return ((inOut[pos].type == inOut[0].type) &&
                  ((inOut[pos].type == nvinfer1::DataType::kFLOAT) || (inOut[pos].type == nvinfer1::DataType::kHALF)));
        }
    }
    else
    {
        return false;
    }
}

void MultiscaleDeformableAttnPlugin::configurePlugin(nvinfer1::DynamicPluginTensorDesc const* inputs, int32_t nbInputs,
    nvinfer1::DynamicPluginTensorDesc const* outputs, int32_t nbOutputs) PLUGIN_NOEXCEPT
{
    // Check for valid input dimensions
    assert(inputs[0].desc.dims.nbDims == 4 && "value nbDims not equal 4");
    assert(inputs[1].desc.dims.nbDims == 2 && "spatial_shape nbDims not equal 2");
    assert(inputs[2].desc.dims.nbDims == 1 && "level_start_index nbDims not equal 1");
    assert(inputs[3].desc.dims.nbDims == 6 && "sampling_location nbDims not equal 6");
    assert(inputs[4].desc.dims.nbDims == 5 && "atten_weight nbDims not equal 5");

    // Check M dimensions consistency
    assert(inputs[0].desc.dims.d[2] == inputs[3].desc.dims.d[2] && "value d2 not equal sampling_location d2");
    assert(inputs[0].desc.dims.d[2] == inputs[4].desc.dims.d[2] && "value d2 not equal atten_weight d2");

    // Check L dimensions consistency
    assert(inputs[1].desc.dims.d[0] == inputs[2].desc.dims.d[0] && "spatial_shape d0 not equal level_start_index d3");
    assert(inputs[1].desc.dims.d[0] == inputs[3].desc.dims.d[3] && "spatial_shape d0 not equal sampling_location d3");
    assert(inputs[1].desc.dims.d[0] == inputs[4].desc.dims.d[3] && "spatial_shape d0 not equal atten_weight d3");

    // Check P dimensions consistency
    assert(inputs[3].desc.dims.d[4] == inputs[4].desc.dims.d[4] && "sampling_location d4 not equal atten_weight d4");

    // Check Lq dimensions consistency
    assert(inputs[3].desc.dims.d[1] == inputs[4].desc.dims.d[1] && "sampling_location d1 not equal atten_weight d1");
}

size_t MultiscaleDeformableAttnPlugin::getWorkspaceSize(nvinfer1::PluginTensorDesc const* inputs, int32_t nbInputs,
    nvinfer1::PluginTensorDesc const* outputs, int32_t nbOutputs) const PLUGIN_NOEXCEPT
{
    return 0;
}

int32_t MultiscaleDeformableAttnPlugin::enqueue(nvinfer1::PluginTensorDesc const* inputDesc,
    nvinfer1::PluginTensorDesc const* outputDesc, void const* const* inputs, void* const* outputs, void* workSpace,
    hipStream_t stream) PLUGIN_NOEXCEPT
{
    int32_t const batch = inputDesc[0].dims.d[0];   
    int32_t spatial_size = inputDesc[0].dims.d[1];
    int32_t num_heads = inputDesc[0].dims.d[2];
    int32_t channels = inputDesc[0].dims.d[3];
    int32_t num_levels = inputDesc[1].dims.d[0];
    int32_t num_query = inputDesc[3].dims.d[1];
    int32_t num_point = inputDesc[3].dims.d[4];
    int32_t rc = 0;
    if (inputDesc[0].type == nvinfer1::DataType::kFLOAT)
    {
        float const* value = static_cast<float const*>(inputs[0]);
        int32_t const* spatialShapes = static_cast<int32_t const*>(inputs[1]);
        int32_t const* levelStartIndex = static_cast<int32_t const*>(inputs[2]);
        float const* samplingLoc = static_cast<float const*>(inputs[3]);
        float const* attnWeight = static_cast<float const*>(inputs[4]);
        float* output = static_cast<float*>(outputs[0]);

        rc = ms_deform_attn_cuda_forward(stream, value, spatialShapes, levelStartIndex, samplingLoc, attnWeight, output,
            batch, spatial_size, num_heads, channels, num_levels, num_query, num_point);
    }
    else if (inputDesc[0].type == nvinfer1::DataType::kHALF)
    {
        const __half* value = static_cast<const __half*>(inputs[0]);
        int32_t const* spatialShapes = static_cast<int32_t const*>(inputs[1]);
        int32_t const* levelStartIndex = static_cast<int32_t const*>(inputs[2]);
        const __half* samplingLoc = static_cast<const __half*>(inputs[3]);
        const __half* attnWeight = static_cast<const __half*>(inputs[4]);
        __half* output = static_cast<__half*>(outputs[0]);
        
        rc = ms_deform_attn_cuda_forward(stream, value, spatialShapes, levelStartIndex, samplingLoc, attnWeight, output,
            batch, spatial_size, num_heads, channels, num_levels, num_query, num_point);
    }

    return rc;
}

void MultiscaleDeformableAttnPlugin::attachToContext(
    cudnnContext* cudnnContext, cublasContext* cublasContext, nvinfer1::IGpuAllocator* gpuAllocator) PLUGIN_NOEXCEPT
{
}

void MultiscaleDeformableAttnPlugin::detachFromContext() PLUGIN_NOEXCEPT {}

// IPluginV2Ext Methods
nvinfer1::DataType MultiscaleDeformableAttnPlugin::getOutputDataType(
    int32_t index, nvinfer1::DataType const* inputTypes, int32_t nbInputs) const PLUGIN_NOEXCEPT
{
    return inputTypes[0];
}

// IPluginV2 Methods
char const* MultiscaleDeformableAttnPlugin::getPluginType() const PLUGIN_NOEXCEPT
{
    return "MultiscaleDeformableAttnPlugin_TRT";
}

char const* MultiscaleDeformableAttnPlugin::getPluginVersion() const PLUGIN_NOEXCEPT
{
    return "1";
}

int32_t MultiscaleDeformableAttnPlugin::getNbOutputs() const PLUGIN_NOEXCEPT
{
    return 1;
}

int32_t MultiscaleDeformableAttnPlugin::initialize() PLUGIN_NOEXCEPT
{
    return 0;
}

void MultiscaleDeformableAttnPlugin::terminate() PLUGIN_NOEXCEPT {}

size_t MultiscaleDeformableAttnPlugin::getSerializationSize() const PLUGIN_NOEXCEPT
{
    return 0;
}

void MultiscaleDeformableAttnPlugin::serialize(void* buffer) const PLUGIN_NOEXCEPT
{
}

void MultiscaleDeformableAttnPlugin::destroy() PLUGIN_NOEXCEPT
{
    delete this;
}

void MultiscaleDeformableAttnPlugin::setPluginNamespace(char const* pluginNamespace) PLUGIN_NOEXCEPT
{
    mNamespace = pluginNamespace;
}
char const* MultiscaleDeformableAttnPlugin::getPluginNamespace() const PLUGIN_NOEXCEPT
{
    return mNamespace.c_str();
}

// Pluginv1 Creator

MultiscaleDeformableAttnPluginCreator::MultiscaleDeformableAttnPluginCreator()
{
    mPluginAttributes.clear();
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

char const* MultiscaleDeformableAttnPluginCreator::getPluginName() const PLUGIN_NOEXCEPT
{
    return "MultiscaleDeformableAttnPlugin_TRT";
}

char const* MultiscaleDeformableAttnPluginCreator::getPluginVersion() const PLUGIN_NOEXCEPT
{
    return "1";
}


nvinfer1::PluginFieldCollection const* MultiscaleDeformableAttnPluginCreator::getFieldNames() PLUGIN_NOEXCEPT
{
    return &mFC;
}

IPluginV2* MultiscaleDeformableAttnPluginCreator::createPlugin(
    char const* name, PluginFieldCollection const* fc) PLUGIN_NOEXCEPT
{
    try
    {
        MultiscaleDeformableAttnPlugin* plugin = new MultiscaleDeformableAttnPlugin();
        return plugin;
    }
    catch (const std::exception& e)
    {
        //caughtError(e);
        std::cerr << e.what() << std::endl;
//        return nullptr;
    }
    return nullptr;
}

IPluginV2* MultiscaleDeformableAttnPluginCreator::deserializePlugin(
    char const* name, void const* serialData, size_t serialLength) PLUGIN_NOEXCEPT
{
    try
    {
        auto plugin = new MultiscaleDeformableAttnPlugin(serialData, serialLength);
        plugin->setPluginNamespace(getPluginNamespace());
        return plugin;
    }
    catch (const std::exception& e)
    {
        std::cerr << e.what() << std::endl;
//        return nullptr;
    }
    return nullptr;
}

void MultiscaleDeformableAttnPluginCreator::setPluginNamespace(char const* pluginNamespace) PLUGIN_NOEXCEPT
{
    mNamespace = pluginNamespace;
}

char const* MultiscaleDeformableAttnPluginCreator::getPluginNamespace() const PLUGIN_NOEXCEPT
{
    return mNamespace.c_str();
}

REGISTER_TENSORRT_PLUGIN(MultiscaleDeformableAttnPluginCreator);
}
